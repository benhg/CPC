/**
 * hello.cu
 * A basic helloworld from CUDA
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * Global means can be called from CPU or GPU
 * Will be run on GPU
 */
__global__ void vec_add(int *a, int *b, int *c){
	*c = *a + *b;
	printf("Hello from thread (%d, %d, %d). We added %d and %d to get %d\n", threadIdx.x, threadIdx.y, threadIdx.z, *a, *b, *c);
} 


int main(){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	// This should be 4 bytes.
	int size = sizeof(int);

	// Alloc device ptrs
	hipMalloc((int**) &d_a, size);
	hipMalloc((void**) &d_b, size);
	hipMalloc((void**) &d_c, size);
	a = 1;
	b = 2;
	c = 0;
	// H2D copies
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	vec_add<<<16,1,1>>>(d_a, d_b, d_c);

	// D2H Copies
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	printf("We added %d and %d to get %d\n", a, b, c);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	return 0;
}
